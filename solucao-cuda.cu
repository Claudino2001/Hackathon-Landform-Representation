
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "omp.h"

#define OUTPUT "output.txt"
#define ANOVA "anova.txt"

/* Landform Sub-Medio Sao Francisco - Petrolina - PNZ */
#define Mxx 5026
#define Mxy 5026
#define INPUT "mde.pnz"

long int n = 0; // Number of dates in the matrix - global variable
double zm;

extern "C"
{
   void dgesv_(int *N, int *NRHS, double *A, int *lda, int *ipiv, double *B, int *ldb, int *info);
}

void dados(double *x, double *y, double *z);
void matrizes(double *A, double *B, double *x, double *y, double *z, int N, int r);
void sistema_Lapack(double *a, double *b, int n);
void calcula(double *a, double *x, double *y, double *z, int r);
void anova(double *a, double *x, double *y, double *z, int N, int r);

int main(int argc, char **argv)
{

   int r = atoi(argv[1]); // Degree of a Polynomial
   int s = r;             // Simplify Polynomial Degree

   int N = (r + 1) * (s + 1); // Number of coefficient of polynomial Landform
   int MAX = (Mxx + 1) * (Mxy + 1);

   double *A = (double *)malloc(sizeof(double) * N * N); /*Dynamic Alocation by column - A[i + j * N] <---------- LAPACK*/
   double *B = (double *)malloc(sizeof(double) * N);
   double *x = (double *)malloc(sizeof(double) * MAX);
   double *y = (double *)malloc(sizeof(double) * MAX);
   double *z = (double *)malloc(sizeof(double) * MAX);

   double t1, t2;

   // printf("\n[Start...]\n\n");

   t1 = omp_get_wtime();

   // printf("\n(Step 1) Extraction of data file mde.pnz \n\n");
   dados(x, y, z);

   // printf("\n(Step 2) Building Matrix \n\n");
   matrizes(A, B, x, y, z, N, r);

   // printf("\n(Step 3) Solver Linear System (Ax=b) ::LAPACK::\n\n");
   sistema_Lapack(A, B, N);

   // printf("\n(Step 4) Reports Landform \n\n");
   anova(B, x, y, z, N, r);

   t2 = omp_get_wtime();

   // printf("\n\n\n[End]\n\n");

   printf("%d\t%5.2f\n", r, t2 - t1);

   free(A);
   free(B);
   free(x);
   free(y);
   free(z);

   return 0;

} /****************************main*************************************************************/

void dados(double *x, double *y, double *z)
{

   int col, row, count = -1;
   FILE *f;
   n = 0;

   if ((f = fopen(INPUT, "r")) == NULL)
   {
      printf("\n Erro I/O\n");
      exit(1);
   }

   for (row = 0; row < Mxy; ++row)
   {
      for (col = 0; col < Mxx; ++col)
      {
         float h;
         int result = fscanf(f, "%f", &h);
         count++;
         if (count % 10 != 0 || h <= 0)
            continue;
         x[n] = row;
         y[n] = col;
         z[n] = h / 2863.0;
         // printf("\n mde[%d,%d] x[%ld]=%f  y[%ld]=%f z[%ld]=%f", row, col, n, x[n], n, y[n], n, z[n]);
         n++;
      }
   }
   // printf("\n n (number of operations for point in the matrix) = %ld\n", n );

   fclose(f);
}

void matrizes(double *A, double *B, double *x, double *y, double *z, int N, int r)
{

   int i, l, c;
   int s = r;

   for (l = 0; l < N; ++l)
   {
      for (c = 0; c < N; ++c)
      {
         A[l + c * N] = 0.0;

         if (c == 0)
            B[l] = 0.0;

         for (i = 0; i < n; ++i)
         {
            A[l + c * N] += pow(x[i], (int)(l / (s + 1)) + (int)(c / (s + 1))) * pow(y[i], l % (r + 1) + c % (r + 1));
            if (c == 0)
               B[l] += z[i] * pow(x[i], (int)(l / (s + 1))) * pow(y[i], l % (r + 1));
         }
      }
   }

   /*
      printf("\nN (size of the matrix A | number of coefficient of polynomial) = %d\n\n", N);

       for (l = 0; l < N; ++l){
         for (c = 0; c < N; ++c)
            printf("%+1.1e ", A[ l + c * N ] );

       printf( "|%+1.1e\n", B[l] );
      }
   */
}

void sistema_Lapack(double *A, double *b, int size)
{

   int NRHS = 1;
   int info;
   int *ipiv = (int *)malloc(sizeof(int) * (10 * size));
   int i;

   dgesv_(&size, &NRHS, A, &size, ipiv, b, &size, &info);

   /*
    if (info != 0)
      printf("[WARNING]: argument had an illegal value\n");
       else
       {
         printf("Solution:\n");

           for (i=0; i < size; ++i)
             printf("[%12g]\n",b[i]);
        }
   */
}

void calcula(double *a, double *x, double *y, double *z, int r)
{
   int i, c, l;
   double Sx, Sy, Sz, e, ze, Se, Sze;
   FILE *f;
   int s = r;

   Sx = Sy = Sz = Se = Sze = 0.0;

   if ((f = fopen(OUTPUT, "w")) == NULL)
   {
      printf("\n Erro I/O");
      exit(3);
   }

   fprintf(f, " ==========================================================\n");
   fprintf(f, "     i      x[i]         y[i]        ye[i]         e[i]   \n");
   fprintf(f, " ==========================================================\n");

   for (i = 0; i < n; i++)
   {
      Sx += x[i];
      Sz += z[i];

      ze = 0.0;

      for (c = 0; c < r + 1; c++)
         for (l = 0; l < s + 1; l++)
            ze += a[c * (r + 1) + l] * pow(x[i], c) * pow(y[i], l);

      e = z[i] - ze;
      Se += e;
      Sze += ze;

      fprintf(f, " %5d    %9.5e    %9.5e    %9.5e    %9.5e\n", i, x[i], y[i], ze, e);
   }

   fprintf(f, " ============================================================\n");
   fprintf(f, " %5ld    %9.5e   %9.5e   %9.5e    %9.5e\n", n, Sx, Sy, Sze, Se);

   zm = Sz / n;

   fclose(f);
}

void anova(double *a, double *x, double *y, double *z, int N, int r)
{
   int i, glReg, glR, glT, c, l;
   double SQReg, SQR, SQT, QMReg, QMR, R2, F, ze;
   FILE *f;
   int s = r;

   SQR = SQReg = 0.0;

   glReg = N;
   glR = n - 2 * N;
   glT = n - N;

   /*
   printf("\n\n-----------------------------------\n");

   for (c = 0; c < r+1; ++c){
       for (l = 0; l < s+1; ++l)
         //printf(" a[%2d,%2d]= %2d\n",c,l, c*(r+1) + l);

   }
   printf("\n\n-----------------------------------\n");
   */

   for (i = 0; i < n; ++i)
   {
      ze = 0.0;
      for (c = 0; c < r + 1; ++c)
         for (l = 0; l < s + 1; ++l)
            ze += a[c + l * (r + 1)] * pow(x[i], c) * pow(y[i], l);

      SQReg += (ze - zm) * (ze - zm);
      SQR += (z[i] - ze) * (z[i] - ze);
   }

   SQT = SQReg + SQR;
   QMReg = SQReg / glReg;
   QMR = SQR / glR;
   F = QMReg / QMR;
   R2 = SQReg / SQT;

   if ((f = fopen(ANOVA, "w")) == NULL)
   {
      printf("\n Error I/O\n");
      exit(2);
   }

   fprintf(f, " \n\n\n\n");
   fprintf(f, " ANOVA\n");
   fprintf(f, " =================================================\n");
   fprintf(f, " FV           gl      SQ         QM          F    \n");
   fprintf(f, " =================================================\n");
   fprintf(f, " Regression  %5d  %12e  %12e  %12e\n", glReg, SQReg, QMReg, F);
   fprintf(f, " Residue     %5d  %12e  %12e      \n", glR, SQR, QMR);
   fprintf(f, " -------------------------------------------------\n");
   fprintf(f, " Total      %5d  %12e            \n", glT, SQT);
   fprintf(f, " =================================================\n");
   fprintf(f, " R^2= %12e                        \n", R2);

   // printf ("P(x)=");

   for (c = 0; c < r + 1; ++c)
      for (l = 0; l < s + 1; ++l)
      {
         fprintf(f, "+x^%d*y^%d*\t%12g\n", c, l, a[c + l * (r + 1)]);
         // printf ("%12g*(x**%d)*(y**%d) + ",a[c + l*(r+1)], c, l );
      }

   fclose(f);
}