#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>  // Para CUDA Runtime

#define Mxx 5026
#define Mxy 5026
#define INPUT "mde.pnz"
#define OUTPUT "output.txt"
#define ANOVA "anova.txt"

// Protótipos das funções
long int n = 0;  // Número global de pontos
double zm = 0.0;  // Média de z

void dados(double *x, double *y, double *z);
void matrizes(double *A, double *B, double *x, double *y, double *z, int N, int r);
void gauss_elimination(double *A, double *B, int N);
void anova(double *A, double *x, double *y, double *z, int N, int r);

__global__ void calculate_matrices_kernel(double *A, double *B, double *x, double *y, double *z, int N, int n, int r) {
    int l = blockIdx.x * blockDim.x + threadIdx.x;  // Índice para linhas de A
    int c = blockIdx.y * blockDim.y + threadIdx.y;  // Índice para colunas de A

    if (l < N && c < N) {
        double local_A = 0.0;
        double local_B = 0.0;

        // Cálculo das matrizes A e B
        for (int i = 0; i < n; ++i) {
            int xIdx = (int)(l / (r + 1)) + (int)(c / (r + 1));
            int yIdx = l % (r + 1) + c % (r + 1);

            local_A += pow(x[i], xIdx) * pow(y[i], yIdx);
            if (c == 0) {
                local_B += z[i] * pow(x[i], xIdx) * pow(y[i], yIdx);
            }
        }

        A[l + c * N] = local_A;
        if (c == 0) {
            B[l] = local_B;
        }
    }
}

int main(int argc, char **argv) {
    int r = atoi(argv[1]);  // Grau do polinômio
    int s = r;               // Simplify Polynomial Degree
    int N = (r + 1) * (s + 1);  // Número de coeficientes de polinômio
    int MAX = (Mxx + 1) * (Mxy + 1);

    double *A = (double*)malloc(sizeof(double) * N * N);
    double *B = (double*)malloc(sizeof(double) * N);
    double *x = (double*)malloc(sizeof(double) * MAX);
    double *y = (double*)malloc(sizeof(double) * MAX);
    double *z = (double*)malloc(sizeof(double) * MAX);

    double t1, t2;

    // Inicializa o tempo
    t1 = omp_get_wtime();

    // Etapas do processo
    dados(x, y, z);

    // Aloca memória na GPU para A, B, x, y, z
    double *d_A, *d_B, *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_A, sizeof(double) * N * N);
    hipMalloc((void **)&d_B, sizeof(double) * N);
    hipMalloc((void **)&d_x, sizeof(double) * n);
    hipMalloc((void **)&d_y, sizeof(double) * n);
    hipMalloc((void **)&d_z, sizeof(double) * n);

    // Copia dados para a memória da GPU
    hipMemcpy(d_x, x, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, sizeof(double) * n, hipMemcpyHostToDevice);

    // Definir blocos e grids para o kernel
    dim3 blockDim(16, 16);  // Blocos de 16x16 threads
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Lançar o kernel para calcular A e B
    calculate_matrices_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_x, d_y, d_z, N, n, r);

    // Sincroniza a execução da GPU
    hipDeviceSynchronize();

    // Copiar os resultados de volta para a CPU
    hipMemcpy(A, d_A, sizeof(double) * N * N, hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, sizeof(double) * N, hipMemcpyDeviceToHost);

    // Libera a memória da GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    // Resolve o sistema linear utilizando eliminação de Gauss
    gauss_elimination(A, B, N);

    // Realiza a análise de variância (ANOVA)
    anova(B, x, y, z, N, r);

    // Finaliza o tempo
    t2 = omp_get_wtime();

    printf("%d\t%5.2f\n", r, t2 - t1);

    // Libera a memória alocada
    free(A);
    free(B);
    free(x);
    free(y);
    free(z);

    return 0;
}

void dados(double *x, double *y, double *z) {
    int col, row, count = -1;
    FILE *f;
    n = 0;  // Inicializa n aqui

    if ((f = fopen(INPUT, "r")) == NULL) {
        printf("\n Erro I/O\n");
        exit(1);
    }

    for (row = 0; row < Mxy; ++row) {
        for (col = 0; col < Mxx; ++col) {
            float h;
            int result = fscanf(f, "%f", &h);
            count++;
            if (count % 10 != 0 || h <= 0) continue;
            x[n] = row;
            y[n] = col;
            z[n] = h / 2863.0;
            n++;  // Incrementa n para o próximo ponto
        }
    }

    // Calcula a média de z para armazenar em zm
    double sum = 0.0;
    for (int i = 0; i < n; i++) {
        sum += z[i];
    }
    zm = sum / n;  // Calcula a média

    fclose(f);
}

void gauss_elimination(double *A, double *B, int N) {
    for (int k = 0; k < N; ++k) {
        // Encontrar o pivô
        double pivot = A[k + k * N];
        if (pivot == 0.0) {
            printf("Pivô nulo encontrado, o sistema é singular.\n");
            exit(1);
        }

        // Normaliza a linha
        for (int j = k + 1; j < N; ++j) {
            A[k + j * N] /= pivot;
        }
        B[k] /= pivot;

        // Elimina os elementos abaixo do pivô
        for (int i = k + 1; i < N; ++i) {
            double factor = A[i + k * N];
            for (int j = k + 1; j < N; ++j) {
                A[i + j * N] -= factor * A[k + j * N];
            }
            B[i] -= factor * B[k];
        }
    }

    // Substituição para trás
    for (int k = N - 1; k >= 0; --k) {
        for (int i = k - 1; i >= 0; --i) {
            double factor = A[i + k * N];
            A[i + k * N] = 0.0;
            B[i] -= factor * B[k];
        }
    }
}

void anova(double *A, double *x, double *y, double *z, int N, int r) {
    int i, glReg, glR, glT, c, l;
    double SQReg, SQR, SQT, QMReg, QMR, R2, F, ze;
    FILE *f;
    int s = r;

    SQR = SQReg = 0.0;
    glReg = N;
    glR = n - 2 * N;
    glT = n - N;

    for (i = 0; i < n; ++i) {  // Usando n aqui
        ze = 0.0;
        for (c = 0; c < r + 1; ++c)
            for (l = 0; l < s + 1; ++l)
                ze += A[c + l * (r + 1)] * pow(x[i], c) * pow(y[i], l);

        SQReg += (ze - zm) * (ze - zm);
        SQR += (z[i] - ze) * (z[i] - ze);
    }

    SQT = SQReg + SQR;
    QMReg = SQReg / glReg;
    QMR = SQR / glR;
    F = QMReg / QMR;
    R2 = SQReg / SQT;

    if ((f = fopen(ANOVA, "w")) == NULL) {
        printf("\n Error I/O\n");
        exit(2);
    }

    fprintf(f, " \n\n\n\n");
    fprintf(f, " ANOVA\n");
    fprintf(f, " =================================================\n");
    fprintf(f, " FV           gl      SQ         QM          F    \n");
    fprintf(f, " =================================================\n");
    fprintf(f, " Regression  %5d  %12e  %12e  %12e\n", glReg, SQReg, QMReg, F);
    fprintf(f, " Residue     %5d  %12e  %12e      \n", glR, SQR, QMR);
    fprintf(f, " -------------------------------------------------\n");
    fprintf(f, " Total      %5d  %12e            \n", glT, SQT);
    fprintf(f, " =================================================\n");
    fprintf(f, " R^2= %12e                        \n", R2);

    for (c = 0; c < r + 1; ++c)
        for (l = 0; l < s + 1; ++l) {
            fprintf(f, "+x^%d*y^%d*\t%12g\n", c, l, A[c + l * (r + 1)]);
        }

    fclose(f);
}
